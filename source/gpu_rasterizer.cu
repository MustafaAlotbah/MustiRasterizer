#include "hip/hip_runtime.h"
/*
	Rasterizer in Cuda C

	Use this way:
	rasterizer.initSession();			<- always include these
	...
	rasterizer.drawLine or whatever ();
	...
	rasterizer.finishSession();			<- always include these

*/

#define DEBUG

#include "gpu_rasterizer.h"





namespace mge {

	/*---------------------CUDA Globals-------------------------*/
	__device__ void* __gpu_buffer = 0;
	__device__ int __gpu_width = 0;
	__device__ int __gpu_height = 0;


	/*---------------------CUDA Functions-------------------------*/
	__global__ void gpuSetBuffer(void* gpuBuffer, int width, int height) {
		__gpu_buffer = gpuBuffer;
		__gpu_width = width;
		__gpu_height = height;
	}

	__device__ void gpuDrawPixel(int x, int y, Pixel p) {
		y = __gpu_height - y - 1;	// correct the orientation
		*((uint32_t*)__gpu_buffer + y * __gpu_width + x) = p.color.value;
	}

	__global__ void gpuClearBufferKernel(Pixel p) {
		gpuDrawPixel(threadIdx.x + blockDim.x * blockIdx.x % (__gpu_width * __gpu_height), __gpu_height -1, p);
	}

	__global__ void gpuDrawPixelKernel(int x, int y, Pixel p) {
		gpuDrawPixel(x, y, p);
	}

	// draws a vertical line
	__global__ void gpuDrawVerticalLine(int x, int yMin, Pixel p) {
		gpuDrawPixel(x, yMin + blockIdx.x * blockDim.x + threadIdx.x, p);
	}

	// draws a horizontal line
	__global__ void gpuDrawHorizontalLine(int y, int xMin, Pixel p) {
		gpuDrawPixel(xMin + blockIdx.x * blockDim.x + threadIdx.x, y, p);
	}

	// with dx > dy (operating on x-axis)
	__global__ void gpuDrawFallingRightLineDx(int xMin, int yMin, float dx , float dy, Pixel p) {
		float y = yMin + (threadIdx.x) * (dy / dx);
		gpuDrawPixel(threadIdx.x + xMin, y, p);
	}

	// with dx < dy (operating on y-axis)
	__global__ void gpuDrawFallingRightLineDy(int xMin, int yMin, float dx, float dy, Pixel p) {
		float x = xMin + (threadIdx.x) * (dx / dy);
		gpuDrawPixel(x, threadIdx.x + yMin, p);
	}


	// with dx > dy (operating on x-axis)
	__global__ void gpuDrawFallingLeftLineDx(int xMin, int yMax, float dx, float dy, Pixel p) {
		float y = yMax - (threadIdx.x) * (dy / dx);
		gpuDrawPixel(threadIdx.x + xMin, y, p);
	}


	// with dx > dy (operating on x-axis)
	__global__ void gpuDrawFallingLeftLineDy(int xMax, int yMin, float dx, float dy, Pixel p) {
		float x = xMax - (threadIdx.x) * (dx / dy);
		gpuDrawPixel(x, threadIdx.x + yMin, p);
	}
	




	/*---------------------Rasterizer Interface-------------------------*/

	GPURasterizer::GPURasterizer(VideoBuffer* buffer) :
	Rasterizer(buffer)
	{
		cudaStatus = hipSetDevice(0);
	}


	GPURasterizer::~GPURasterizer(){
	}


	bool GPURasterizer::clearBuffer(Pixel p) {

		dim3 thrds(1000, 1);
		dim3 blcks(lastAllocatedSize / 1000 + 1, 1);
		gpuClearBufferKernel <<<blcks, thrds >>> (p);
		return true;
	}


	bool GPURasterizer::initSession(Pixel p) {
		lastAllocatedSize = buffer->height * buffer->width;
		// allocate GPU Memory (will be disposed of by finishing the session
		hipMalloc((void**)&gpuBuffer, sizeof(uint32_t) * lastAllocatedSize) ;
		gpuSetBuffer << <1, 1 >> > (gpuBuffer, buffer->width, buffer->height);
		clearBuffer(p);
		return true;
	}


	bool GPURasterizer::finishSession() {
		// copy GPU buffer into CPU buffer and free GPU buffer
		hipMemcpy(buffer->addr, gpuBuffer, lastAllocatedSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
		return hipFree(gpuBuffer) == hipSuccess;
	}



	/*
	Pixel has its x, y, z and depth information
	Note the the pixel has to be in the screen space already!
	!!! not suited for a single used!!!
	*/

	bool GPURasterizer::drawPixel(int x, int y, Pixel p) {
		gpuDrawPixelKernel<<<1, 1>>>(x, y, p);
		return true;
	}


	// This line looks like this ( | )
	bool GPURasterizer::drawVerticalLine(int x, int y1, int y2, Pixel p) {
		int startY = min(min(y1, y2), buffer->height);
		int endY = max(max(y1, y2), 0);
		uint32_t numPixels = endY + startY;
		gpuDrawVerticalLine<<<1, numPixels>>>(x, startY, p);

		return true;
	}


	// This line looks like this ( - )
	bool GPURasterizer::drawHorizontalLine(int y, int x1, int x2, Pixel p) {
		int startX = min(min(x1, x2), buffer->width);
		int endX = max(max(x1, x2), 0);
		uint32_t numPixels = endX + startX;
		gpuDrawHorizontalLine << <1, numPixels >> > (y, startX, p);
		return true;
	}


	// this line looks like ( \ )
	bool GPURasterizer::drawFallingRightLine(int x1, int y1, int x2, int y2, Pixel p) {

		int minX = min(min(x1, x2), buffer->width);
		int maxX = max(max(x1, x2), 0);

		int minY = min(min(y1, y2), buffer->height);
		int maxY = max(max(y1, y2), 0);

		float dx = maxX - minX;
		float dy = maxY - minY;

		if (dx >= dy) 
			gpuDrawFallingRightLineDx <<<1, dx>>> (minX, minY, dx, dy, p);
		else 
			gpuDrawFallingRightLineDy <<<1, dy >>> (minX, minY, dx, dy, p);


		return true;
	}


	// this line looks like ( / )
	bool GPURasterizer::drawFallingLeftLine(int x1, int y1, int x2, int y2, Pixel p) {

		int xMin = min(min(x1, x2), buffer->width);
		int xMax = max(max(x1, x2), 0);

		int yMin = min(min(y1, y2), buffer->height);
		int yMax = max(max(y1, y2), 0);

		float dx = xMax - xMin;
		float dy = yMax - yMin;


		if (dx >= dy)
			gpuDrawFallingLeftLineDx << <1, dx >> > (xMin, yMax, dx, dy, p);
		else
			gpuDrawFallingLeftLineDy << <1, dy >> > (xMax, yMin, dx, dy, p);

		return true;
	}











}